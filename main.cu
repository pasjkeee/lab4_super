#include <netdb.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/socket.h>
#include <unistd.h>
#include <arpa/inet.h>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define MAX 1000
#define PORT 8080
#define SA struct sockaddr

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void matrixCalc(int *a, int *res, int N) {

    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < N) {
        int sum = 0;

        for (int k = 0; k < N; ++k) {
            int el = a[col * N + k];
            sum += el;
        }

        res[col] = sum / N;
    }


}

int counter(int res_buf, int * submit, int * res_arr, int ** arr) {

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    float res_time = 0;
    time_t start = clock();
    printf("INFO: Start counting \n");

    int *a, *res, *h_a;

    size_t bytes = res_buf * res_buf * sizeof(int);
    size_t bytes_res = res_buf * sizeof(int);

    h_a = (int*) malloc(bytes);

    for (int i=0; i < res_buf; i++) {
        for (int j=0; j < res_buf; j++) {
            h_a[i*res_buf + j] = arr[i][j];
            if (h_a[i*res_buf + j] == 0) {
                h_a[i*res_buf + j] = rand()%100;
            }
        }
    }

    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&a), bytes));
    gpuErrchk(hipMemcpyAsync(a, h_a, bytes, hipMemcpyHostToDevice));
    gpuErrchk(hipMalloc(reinterpret_cast<void **>(&res), bytes));
    int block_size = 1024;
    int grid_size = (res_buf + block_size) / block_size;

    dim3 DimGrid(grid_size, 1,1);
    dim3 DimBlock(block_size, 1,1);

    matrixCalc<<<DimGrid,DimBlock>>>(a, res, res_buf);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk( hipMemcpyAsync(res_arr, res, res_buf, hipMemcpyDeviceToHost));

    time_t end = clock();

    res_time = ((float)(end - start) / 1000000.0F ) * 1000;
    printf("INFO: Stop counting \n");

    gpuErrchk(hipFree( res ));
    gpuErrchk(hipFree( a ));
    return res_time;
}

void end_transaction (int res_buf, int * submit, int * res_arr, int ** arr, int sockfd) {
    char buf[MAX];
    float res_time = counter(res_buf, submit, res_arr, arr);

    for (int i = 0; i < res_buf; i++) {
        sprintf(buf, "%d \n", res_arr[i]);
        write(sockfd, buf, sizeof(buf));
    }

    sprintf(buf, "%d", *submit / 1024);
    write(sockfd, buf, sizeof(buf));
    sprintf(buf, "%f", res_time);
    write(sockfd, buf, sizeof(buf));
    write(sockfd, "exit", sizeof("exit"));
}

int get_res_buf(char * val) {
    printf("INFO: Принятие результирующего буффера => выделение памяти %d \n", atoi(val));
    return atoi(val);
}

void func(int sockfd)
{
    char buff[MAX];
    int n;
    for (;;) {
        bzero(buff, sizeof(buff));
        printf("Enter the string from client: ");
        write(sockfd, "start", sizeof("start"));
        bzero(buff, sizeof(buff));
        read(sockfd, buff, sizeof("start"));
        printf("INFO: read %s \n", buff);

        if ((strncmp(buff, "start", sizeof("start"))) == 0) {
            int submit = 0;
            int** arr;
            int* res_arr;
            int res_buf = 0;
            int index_row = 0;
            int index_col = 0;

            printf("INFO: Start from server reciever \n");
            char *val = (char*)malloc(sizeof(buff));

            while ((strncmp(buff, "end", sizeof("end"))) != 0) {
                read(sockfd, buff, sizeof(buff));


                for (int i = 0; i < strlen(buff); i++) {


                    if (buff[i] == 's' || buff[i] == 't' ||buff[i] == 'a' ||buff[i] == 'r') {
                        continue;
                    }

                    if(buff[i] == 'e') {
                        memset(val, 0, sizeof(buff));
                        end_transaction(res_buf, &submit, res_arr, arr, sockfd);
                        return;
                    }

                    if (buff[i] == ' ') {
                        arr[index_row][index_col] = atoi(val);
                        memset(val, 0, sizeof(buff));
                        index_col++;
                        continue;
                    }

                    if (buff[i] == '\n') {

                        if (res_buf == 0) {
                            res_buf = get_res_buf(val);

                            res_arr = (int*) malloc(res_buf * sizeof(int));
                            arr = (int**) malloc(res_buf * sizeof(int*));
                            for (int j = 0; j < res_buf; j++) {
                                arr[j] = (int*) malloc(res_buf * sizeof(int));
                            }
                            memset(val, 0, sizeof(buff));
                            continue;
                        }

                        arr[index_row][index_col] = atoi(val);
                        memset(val, 0, sizeof(buff));
                        index_row++;
                        index_col = 0;
                        continue;
                    }

                    if (buff[i] != '\t'){
                        sprintf(val, "%s%c", val, buff[i]);
                    }
                }

            }
            continue;
        }

        printf("From Server : %s", buff);
        if ((strncmp(buff, "exit", 4)) == 0) {
            printf("Client Exit...\n");
            break;
        }
    }
}

int create_and_verify_socket() {
    int sock_fd = socket(AF_INET, SOCK_STREAM, 0);
    if (sock_fd == -1) {
        printf("ERROR: Socket creation failed...\n");
        exit(0);
    }
    printf("INFO: Socket successfully created..\n");
    return sock_fd;
}

int configure_serv_addr(struct sockaddr_in * serv_addr, int sockfd) {
    bzero(serv_addr, sizeof(*serv_addr));

    serv_addr->sin_family = AF_INET;
    serv_addr->sin_addr.s_addr = inet_addr("127.0.0.1");
    serv_addr->sin_port = htons(PORT);

    // Binding newly created socket to given IP and verification
    if ((connect(sockfd, (SA*)serv_addr, sizeof(*serv_addr))) != 0) {
        printf("ERROR: Socket bind failed...\n");
        exit(0);
    }
    else
        printf("INFO: Socket successfully binded..\n");

    return sockfd;
}

//сформировать результирующий вектор как среднее по каждой строке исходной квадратной матрицы
int main(int argc, char **argv) {

    int sockfd;
    struct sockaddr_in serv_addr;

    sockfd = create_and_verify_socket();
    configure_serv_addr(&serv_addr, sockfd);

    // main func
    func(sockfd);

    close(sockfd);
}
